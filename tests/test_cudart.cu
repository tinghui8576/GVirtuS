#include "hip/hip_runtime.h"
/*
 * Written By: Theodoros Aslanidis <theodoros.aslanidis@ucdconnect.ie>
 *             School of Computer Science, University College Dublin
 */

#include <hip/hip_runtime.h> /* cuuint64_t */
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

#define CUDA_CHECK(err) ASSERT_EQ((err), hipSuccess)

__device__ int intDeviceVariable = 0;

TEST(cudaRT, getDeviceCount) {
    int count = 0;
    CUDA_CHECK(hipGetDeviceCount(&count));
    ASSERT_GT(count, 0);
}

TEST(cudaRT, ThreadExchangeStreamCaptureMode) {
    hipStreamCaptureMode original_mode = hipStreamCaptureModeThreadLocal;
    hipStreamCaptureMode previous_mode;

    // Exchange thread-local with current mode, original_mode gets overwritten with previous
    previous_mode = original_mode;
    CUDA_CHECK(hipThreadExchangeStreamCaptureMode(&original_mode));

    // Ensure that exchange actually happened: value at `original_mode` now holds the previous
    ASSERT_NE(previous_mode, original_mode);

    // Now push the original mode back to restore thread state
    CUDA_CHECK(hipThreadExchangeStreamCaptureMode(&original_mode));

    // Ensure that the original mode is restored
    ASSERT_EQ(original_mode, previous_mode);
}

TEST(cudaRT, MemPoolGetAttribute) {
    hipMemPool_t memPool;
    CUDA_CHECK(hipDeviceGetDefaultMemPool(&memPool, 0));

    cuuint64_t threshold = 0;
    CUDA_CHECK(hipMemPoolGetAttribute(memPool, hipMemPoolAttrReleaseThreshold, &threshold));
}

TEST(cudaRT, MallocFree) {
    void* devPtr = nullptr;
    CUDA_CHECK(hipMalloc(&devPtr, 1024));
    CUDA_CHECK(hipFree(devPtr));
}

TEST(cudaRT, MemcpySync) {
    int h_src = 42;
    int h_dst = 0;
    int* d_ptr;
    CUDA_CHECK(hipMalloc(&d_ptr, sizeof(int)));

    CUDA_CHECK(hipMemcpy(d_ptr, &h_src, sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(&h_dst, d_ptr, sizeof(int), hipMemcpyDeviceToHost));
    ASSERT_EQ(h_dst, 42);

    CUDA_CHECK(hipFree(d_ptr));
}

TEST(cudaRT, MemcpyAsync) {
    int h_src = 24;
    int h_dst = 0;
    int* d_ptr;
    hipStream_t stream;
    CUDA_CHECK(hipMalloc(&d_ptr, sizeof(int)));
    CUDA_CHECK(hipStreamCreate(&stream));

    CUDA_CHECK(hipMemcpyAsync(d_ptr, &h_src, sizeof(int), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(&h_dst, d_ptr, sizeof(int), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));
    ASSERT_EQ(h_dst, 24);

    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipFree(d_ptr));
}

TEST(cudaRT, Memset) {
    int* d_ptr;
    CUDA_CHECK(hipMalloc(&d_ptr, sizeof(int)));
    CUDA_CHECK(hipMemset(d_ptr, 0, sizeof(int)));

    int h_val = 1;
    CUDA_CHECK(hipMemcpy(&h_val, d_ptr, sizeof(int), hipMemcpyDeviceToHost));
    ASSERT_EQ(h_val, 0);

    CUDA_CHECK(hipFree(d_ptr));
}

TEST(cudaRT, StreamCreateDestroySynchronize) {
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipStreamDestroy(stream));
}

TEST(cudaRT, StreamCaptureBeginEnd) {
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    hipStreamCaptureMode mode = hipStreamCaptureModeThreadLocal;
    CUDA_CHECK(hipStreamBeginCapture(stream, mode));
    hipGraph_t graph;
    CUDA_CHECK(hipStreamEndCapture(stream, &graph));
    CUDA_CHECK(hipStreamDestroy(stream));
}

TEST(cudaRT, GraphCreateDestroy) {
    hipGraph_t graph;
    CUDA_CHECK(hipGraphCreate(&graph, 0));
    CUDA_CHECK(hipGraphDestroy(graph));
}

__global__ void dummyKernel() {
}

TEST(cudaRT, GraphInstantiateDestroy) {
    hipGraph_t graph;
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    hipStreamCaptureMode mode = hipStreamCaptureModeThreadLocal;
    hipGraphNode_t* nodes = NULL;
    size_t numNodes = 0;
    CUDA_CHECK(hipStreamBeginCapture(stream, mode));
    dummyKernel<<<1, 1, 0, stream>>>(); 
    CUDA_CHECK(hipStreamEndCapture(stream, &graph));
    CUDA_CHECK(hipGraphGetNodes(graph, nodes, &numNodes));
    ASSERT_EQ(numNodes, 1);
    hipGraphExec_t graphExec;
    CUDA_CHECK(hipGraphInstantiate(&graphExec, graph, 0));
    CUDA_CHECK(hipGraphDestroy(graph));
    CUDA_CHECK(hipGraphExecDestroy(graphExec));
    CUDA_CHECK(hipStreamDestroy(stream));
}

TEST(cudaRT, GraphLaunch) {
    hipGraph_t graph;
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    hipStreamCaptureMode mode = hipStreamCaptureModeThreadLocal;
    hipGraphNode_t* nodes = NULL;
    size_t numNodes = 0;
    CUDA_CHECK(hipStreamBeginCapture(stream, mode));
    dummyKernel<<<1, 1, 0, stream>>>(); 
    CUDA_CHECK(hipStreamEndCapture(stream, &graph));
    CUDA_CHECK(hipGraphGetNodes(graph, nodes, &numNodes));
    hipGraphExec_t graphExec;
    CUDA_CHECK(hipGraphInstantiate(&graphExec, graph, 0));
    CUDA_CHECK(hipGraphDestroy(graph));
    CUDA_CHECK(hipGraphLaunch(graphExec, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));
}

TEST(cudaRT, GetDevice) {
    int device;
    CUDA_CHECK(hipGetDevice(&device));
}

TEST(cudaRT, SetDevice) {
    int device = 0;
    CUDA_CHECK(hipSetDevice(device));
}

TEST(cudaRT, DeviceSynchronize) { CUDA_CHECK(hipDeviceSynchronize()); }

__global__ void simpleKernel(int* output) { *output = 123; }

TEST(cudaRT, LaunchKernel) {
    int* d_output;
    CUDA_CHECK(hipMalloc(&d_output, sizeof(int)));
    CUDA_CHECK(hipMemset(d_output, 0, sizeof(int)));

    void* args[] = {&d_output};

    dim3 grid(1), block(1);
    CUDA_CHECK(hipLaunchKernel((const void*)simpleKernel, grid, block, args, 0, nullptr));

    int h_output = 0;
    CUDA_CHECK(hipMemcpy(&h_output, d_output, sizeof(int), hipMemcpyDeviceToHost));
    ASSERT_EQ(h_output, 123);

    CUDA_CHECK(hipFree(d_output));
}

TEST(cudaRT, PushCallConfiguration) {
    dim3 grid(1), block(1);
    size_t shared = 0;
    hipStream_t stream = 0;
    CUDA_CHECK(__cudaPushCallConfiguration(grid, block, shared, stream));
}

TEST(CudaRT, KernelLaunchWithTripletSyntax) {
    int* d_out = nullptr;
    int h_out = 0;

    // Allocate memory on device
    CUDA_CHECK(hipMalloc(&d_out, sizeof(int)));

    // Launch kernel with <<<>>> syntax
    simpleKernel<<<1, 1>>>(d_out);

    // Wait for kernel to complete
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy result back to host
    CUDA_CHECK(hipMemcpy(&h_out, d_out, sizeof(int), hipMemcpyDeviceToHost));

    // Verify kernel result
    ASSERT_EQ(h_out, 123);

    CUDA_CHECK(hipFree(d_out));
}

TEST(cudaRT, EventCreateRecordSynchronizeElapsedTimeDestroy) {
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    CUDA_CHECK(hipEventRecord(start));
    CUDA_CHECK(hipEventRecord(stop));

    CUDA_CHECK(hipEventSynchronize(stop));

    float elapsed_ms = 0;
    CUDA_CHECK(hipEventElapsedTime(&elapsed_ms, start, stop));
    ASSERT_GT(elapsed_ms, 0.0f);

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
}
